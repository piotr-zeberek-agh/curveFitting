// System includes
#include <stdio.h>
#include <assert.h>
#include <chrono>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//matrix operation using GPU
#include "matrixOperationsGPU.cuh"

#define BLOCK_SIZE 32

void inverseMatrixGPU(double *A, double *inv_A, int size);

template<typename T>
void printMatrix(T *A, int rows, int cols);

template<typename T>
void printDeviceMatrix(T* d_A, int rows, int cols);

int main(){
	
  dim3 threads(BLOCK_SIZE,BLOCK_SIZE,1);
  
  int nSamples=11;
  int order=12;
  
  //Stream for synchronization and timing 
  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  
  //Allocate Host x,y and B vectors
  double *h_x, *h_y, *h_B;
  
  unsigned int mem_size_samples = sizeof(double) * nSamples;
  checkCudaErrors(hipHostMalloc(&h_x, mem_size_samples));
  checkCudaErrors(hipHostMalloc(&h_y, mem_size_samples));
  
  unsigned int mem_size_B = sizeof(double) * (order + 1);
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  
  //Put some values into B and print
  for(int i=0; i<order+1; i++){
  	h_B[i]=i%3;
  }
  
  printf("B = ");
  printMatrix(h_B,1,order+1);
  
  //Allocate Device x,y and B vectors
  double *d_x, *d_y, *d_B;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_x), mem_size_samples));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_y), mem_size_samples));
  
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
   
  //Copy x,y and B from Host to Device
  checkCudaErrors(hipMemcpyAsync(d_x, h_x, mem_size_samples, hipMemcpyHostToDevice, stream));
  checkCudaErrors(hipMemcpyAsync(d_y, h_y, mem_size_samples, hipMemcpyHostToDevice, stream));
  
  checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));
  
  //Generate range of x and print
  double start = -5.0f;
  double stop = 5.0f;
  size_t xInitBlocks = (nSamples + BLOCK_SIZE - 1) / BLOCK_SIZE;
  
  xInitRange<<<xInitBlocks, BLOCK_SIZE, 0, stream>>>(d_x,start,stop,nSamples);
  checkCudaErrors(hipStreamSynchronize(stream));
  
  printf("x = ");
  printDeviceMatrix(d_x,1,nSamples);

  //Allocate host Vandermonde matrix 
  dim3 dimsV(order+1,nSamples,1);
  
  double *h_V;
  unsigned int mem_size_V = sizeof(double) * dimsV.x * dimsV.y;
  checkCudaErrors(hipHostMalloc(&h_V, mem_size_V));
  
  //Allocate device Vandermonde matrix
  double *d_V;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_V), mem_size_V));
  
  //Copy Vandermonde matrix to Device
  checkCudaErrors(hipMemcpyAsync(d_V, h_V, mem_size_V, hipMemcpyHostToDevice, stream));
  
  //Initialize Vandermonde matrix
  dim3 blocksVandermonde((dimsV.x + BLOCK_SIZE - 1) / BLOCK_SIZE, (dimsV.y + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
  Vandermonde<<<blocksVandermonde, threads, 0, stream>>>(d_x, d_V, order, nSamples);
  checkCudaErrors(hipStreamSynchronize(stream));
  
  //Calculate y=V*B and print
  dim3 blocksY((nSamples + BLOCK_SIZE - 1) / BLOCK_SIZE, (dimsV.y + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
  matMul<<<blocksY,threads>>>(d_V, d_B, d_y, dimsV.y, dimsV.x, 1);
  checkCudaErrors(hipStreamSynchronize(stream));
  
  printf("y = ");
  printDeviceMatrix(d_y,1,nSamples);
  
  checkCudaErrors(hipHostFree(h_x));
  checkCudaErrors(hipHostFree(h_y));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_V));
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_V));
}

void inverseMatrixGPU(double *A, double *inv_A, int size){

	hipStream_t stream;
	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  
	//Allocate identity and auxiliary matrix on device
	int mem_size = size * size * sizeof(double);
	double *d_I, *d_Aux, *d_ref, *d_ref2;
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_I), mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_Aux), mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_ref), mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_ref2), mem_size));
	
	checkCudaErrors(hipMemcpyAsync(d_ref, A, mem_size, hipMemcpyDeviceToDevice, stream));
	checkCudaErrors(hipStreamSynchronize(stream));
	
	//Initialize identity matrix
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE,1);
	dim3 blocks((size + BLOCK_SIZE - 1) / BLOCK_SIZE, (size + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
	
	initIdentityMatrix<<<blocks, threads, 0, stream>>>(d_I,size);
	checkCudaErrors(hipStreamSynchronize(stream));
	
	for(int i=0;i<size;i++){
		reduceRow<<<blocks, threads, 0, stream>>>(d_ref,d_I,d_Aux,size,i);
		reduceRow<<<blocks, threads, 0, stream>>>(d_ref,d_ref,d_ref2,size,i);

		substractRow<<<blocks, threads, 0, stream>>>(d_ref2,d_Aux,d_I,size,i);
		substractRow<<<blocks, threads, 0, stream>>>(d_ref2,d_ref2,d_ref,size,i);
	}
	checkCudaErrors(hipStreamSynchronize(stream));
	
	checkCudaErrors(hipMemcpyAsync(inv_A, d_I, mem_size, hipMemcpyDeviceToDevice, stream));
	checkCudaErrors(hipStreamSynchronize(stream));
	
	//printf("inverse matrix\n");
	//printDeviceMatrix(inv_A,size,size);
  
	checkCudaErrors(hipFree(d_I));
	checkCudaErrors(hipFree(d_Aux));
	checkCudaErrors(hipFree(d_ref));
	checkCudaErrors(hipFree(d_ref2));
  
}

template<typename T>
void printMatrix(T *A, int rows, int cols){

	for(int i=0;i<rows;i++){
		for(int j=0;j<cols;j++){
			std::cout<<A[i*cols+j]<<" ";
		}
		
		printf("\n");
	}
}

template<typename T>
void printDeviceMatrix(T* d_A, int rows, int cols){
	
	T *temp;
	int mem_size = rows * cols * sizeof(T);
	
	checkCudaErrors(hipHostMalloc(&temp, mem_size));
	checkCudaErrors(hipMemcpyAsync(temp, d_A, mem_size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipDeviceSynchronize());
	
	printMatrix(temp, rows, cols);
	
	checkCudaErrors(hipHostFree(temp));
}
