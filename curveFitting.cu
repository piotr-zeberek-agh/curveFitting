// System includes
#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <cmath>
#include <iomanip>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//matrix operation using GPU
#include "matrixOperationsGPU.cuh"

#define BLOCK_SIZE 32

void invertMatrixGPU(double* A, int size);

void readData(double *x, double *y, const char *file_name, int nSamples);

void demo(int nSamples, int order);

void regression(double* x, double* y, int nSamples, int order, double* d_V=nullptr);

template<typename T>
void printMatrix(T *A, dim3 dims);

template<typename T>
void printDeviceMatrix(T* d_A, dim3 dims);

int main(int argc, char *argv[]){
	int nSamples=0;
	int order=0;
	size_t mem_size_samples = 0;
	const char *file_name = nullptr;
	
	
	switch(argc){
		case 1:	nSamples=64;
			order=2;
			printf("Running demo with %d samples and order of %d\n",nSamples,order);
		  	demo(nSamples,order);
		  	break;
		  	
		case 4:	//Read execution parameters
			file_name = argv[1];
			nSamples = atoi(argv[2]);
			order = atoi(argv[3]);
			printf("Polynomial regression for (x,y) pairs from \"%s\", number of samples: %d, order of polynomial: %d\n",file_name,nSamples,order);
			
			//Allocate Host x,y vector
			double *h_x, *h_y;
			mem_size_samples = nSamples * sizeof(double);
			checkCudaErrors(hipHostMalloc(&h_x, mem_size_samples));
			checkCudaErrors(hipHostMalloc(&h_y, mem_size_samples));
			
			//Read data from file
			readData(h_x, h_y, file_name, nSamples);

			//Perform regression
			regression(h_x, h_y, nSamples, order);
			break;
			
		default: printf("Wrong execution\n");
	}
}


void demo(int nSamples, int order){

  dim3 threads(BLOCK_SIZE,BLOCK_SIZE,1);
  
  //Stream for synchronization
  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  
  //Allocate Host B vector
  double *h_B;
  dim3 dimsB(1,order+1,1);
  size_t mem_size_B = dimsB.x * dimsB.y * sizeof(double);
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  
  //Put some values into B
  for(int i=0; i<order+1; i++){
  	h_B[i]=i%3;
  }
  
  printf("B:\n");
  for(int i=0; i<dimsB.y; i++){
  	printf("%6.4f ",h_B[i]);
  }
  printf("\n");
  
  //Allocate Device x, y, and B vectors
  size_t mem_size_samples = sizeof(double) * nSamples;
  double *d_x, *d_y, *d_B;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_x), mem_size_samples));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_y), mem_size_samples));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));

  //Copy Host B into Device B
  checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

  //Generate range of x
  double start = -5.0f;
  double stop = 5.0f;
  size_t xInitBlocks = (nSamples + BLOCK_SIZE - 1) / BLOCK_SIZE;
  
  xInitRange<<<xInitBlocks, BLOCK_SIZE, 0, stream>>>(d_x,start,stop,nSamples);
  
  //Allocate device Vandermonde matrix
  double *d_V;
  dim3 dimsV(order+1,nSamples,1);
  size_t mem_size_V = sizeof(double) * dimsV.x * dimsV.y;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_V), mem_size_V));
  
  //Initialize Vandermonde matrix
  dim3 blocksVandermonde((dimsV.x + BLOCK_SIZE - 1) / BLOCK_SIZE, (dimsV.y + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
  Vandermonde<<<blocksVandermonde, threads, 0, stream>>>(d_x, d_V, order, nSamples);
  
  //Calculate y=V*B
  dim3 blocksY(1, (dimsV.y + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
  matMulGPU<<<blocksY,threads,0,stream>>>(d_V, d_B, d_y, dimsV.y, dimsV.x, 1);
  checkCudaErrors(hipStreamSynchronize(stream));
      
  //Perform regression
  regression(d_x,d_y,nSamples,order,d_V);
  
  //Free memory
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipFree(d_B));
}

void regression(double* x, double* y, int nSamples, int order, double* d_V){

	//Stream for synchronization
	hipStream_t stream;
	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));\
	
	double *d_x, *d_y;
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE,1);
	
	dim3 dimsV(order+1,nSamples,1);
	size_t mem_size_V = sizeof(double) * dimsV.x * dimsV.y;
	dim3 blocksVandermonde((dimsV.x + BLOCK_SIZE - 1) / BLOCK_SIZE, (dimsV.y + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
	
	if(!d_V){
		//Allocate Device x, y
		size_t mem_size_samples = sizeof(double) * nSamples;
		
		checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_x), mem_size_samples));
		checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_y), mem_size_samples));
		
		//Copy data from Host to Device x,y
  		checkCudaErrors(hipMemcpyAsync(d_x, x, mem_size_samples, hipMemcpyHostToDevice, stream));
  		checkCudaErrors(hipMemcpyAsync(d_y, y, mem_size_samples, hipMemcpyHostToDevice, stream));
		
		//Allocate device Vandermonde matrix
		checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_V), mem_size_V));
		  
		//Initialize Vandermonde matrix
		Vandermonde<<<blocksVandermonde, threads, 0, stream>>>(d_x, d_V, order, nSamples);
		
	}else{
		d_x=x;
		d_y=y;
	}
	
  //Allocate device transposed Vandermonde matrix
  double *d_V_T;
  dim3 dimsV_T(nSamples,order+1,1);
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_V_T), mem_size_V));
  
  //Transpose Vandermonde matrix
  transposeGPU<<<blocksVandermonde,threads,0,stream>>>(d_V,d_V_T,dimsV.y,dimsV.x);
  
  //Allocate device matrix temp for storing temporary outcome
  //temp = V_T * V
  double *d_temp;
  size_t mem_size_temp = dimsV_T.y * dimsV.x * sizeof(double);
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_temp), mem_size_temp));
  
  //Calculate temp = V_T * V
  dim3 blocksTemp((dimsV.x + BLOCK_SIZE - 1) / BLOCK_SIZE, (dimsV_T.y + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
  matMulGPU<<<blocksTemp,threads,0,stream>>>(d_V_T,d_V,d_temp,dimsV_T.y,dimsV_T.x,dimsV.x);
  
  //Synchronize before calling matrix inverting function
  checkCudaErrors(hipStreamSynchronize(stream));
  
  //Invert matrix temp = V_T * V
  invertMatrixGPU(d_temp,dimsV.x);
  
  //Allocate device matrix temp2 for storing temporary outcome
  //temp2 = temp * V_T
  double *d_temp2;
  size_t mem_size_temp2 = dimsV.x * dimsV_T.x * sizeof(double);
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_temp2), mem_size_temp2));
  
  //Calculate temp2 = temp * V_T
  dim3 blocksTemp2((dimsV_T.x + BLOCK_SIZE - 1) / BLOCK_SIZE, (dimsV.x + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
  matMulGPU<<<blocksTemp2,threads,0,stream>>>(d_temp,d_V_T,d_temp2,dimsV.x,dimsV.x,dimsV_T.x);
  
  //Allocate Device B_est vector
  double *d_B_est;
  dim3 dimsB(1,order+1,1);
  size_t mem_size_B = sizeof(double) * (order+1);
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B_est), mem_size_B));
  
  //Calculate B_est = temp2 * y
  dim3 blocksB(1, (dimsV.x + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
  matMulGPU<<<blocksB,threads,0,stream>>>(d_temp2,d_y,d_B_est,dimsV.x,nSamples,1);
  
  //Allocate Host B_est vector
  double *h_B_est;
  checkCudaErrors(hipHostMalloc(&h_B_est, mem_size_B));

  //Copy data from Device to Host B_est
  checkCudaErrors(hipMemcpyAsync(h_B_est, d_B_est, mem_size_B, hipMemcpyDeviceToHost, stream));
  
  //Synchronize before retriving results calling matrix printing function
  checkCudaErrors(hipStreamSynchronize(stream));

  //Print B_est
  printf("B_est:\n");
  for(int i=0; i<dimsB.y; i++){
  	printf("%6.4f ",h_B_est[i]);
  }
  printf("\n");
	
  //Free memory
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipHostFree(h_B_est));
  checkCudaErrors(hipFree(d_B_est));
  checkCudaErrors(hipFree(d_V));
  checkCudaErrors(hipFree(d_V_T));
  checkCudaErrors(hipFree(d_temp));
  checkCudaErrors(hipFree(d_temp2));
}

void invertMatrixGPU(double *A, int size){

	hipStream_t stream;
	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  
	//Allocate reference and temporary matrices on device
	size_t mem_size = size * size * sizeof(double);
	double *d_temp, *d_ref, *d_ref2;
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_temp), mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_ref), mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_ref2), mem_size));
	
	//Copy data from matrix A to reference matrix
	checkCudaErrors(hipMemcpyAsync(d_ref, A, mem_size, hipMemcpyDeviceToDevice, stream));
	
	//Initialize Aux as identity matrix
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE,1);
	dim3 blocks((size + BLOCK_SIZE - 1) / BLOCK_SIZE, (size + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
	
	initIdentityMatrix<<<blocks, threads, 0, stream>>>(A,size);
	
	//Perform matrix inversion using Gauss-Jordan elimination
	for(int i=0;i<size;i++){
		reduceRow<<<blocks, threads, 0, stream>>>(d_ref,A,d_temp,size,i);
		reduceRow<<<blocks, threads, 0, stream>>>(d_ref,d_ref,d_ref2,size,i);

		substractRow<<<blocks, threads, 0, stream>>>(d_ref2,d_temp,A,size,i);
		substractRow<<<blocks, threads, 0, stream>>>(d_ref2,d_ref2,d_ref,size,i);
	}
	
	//Synchronize output
	checkCudaErrors(hipStreamSynchronize(stream));
  
  	//Free memory
	checkCudaErrors(hipFree(d_temp));
	checkCudaErrors(hipFree(d_ref));
	checkCudaErrors(hipFree(d_ref2));
}

void readData(double *x, double *y, const char *file_name, int nSamples){

	std::ifstream file(file_name);

	if ( file.is_open() ){

		for(int i=0; i<nSamples; i++){
			file >> x[i] >> y[i];
		}
	}else{
		printf("Cannot read file %s",file_name);
	}	

}

template<typename T>
void printMatrix(T *A, dim3 dims){

	for(int i=0;i<dims.y;i++){
		for(int j=0;j<dims.x;j++){
			std::cout<<std::setprecision(20)<<A[i*dims.x+j]<<" ";
		}
		
		printf("\n");
	}
}

template<typename T>
void printDeviceMatrix(T* d_A, dim3 dims){
	
	T *temp;
	size_t mem_size = dims.x * dims.y * sizeof(T);
	
	checkCudaErrors(hipHostMalloc(&temp, mem_size));
	checkCudaErrors(hipMemcpyAsync(temp, d_A, mem_size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipDeviceSynchronize());
	
	printMatrix(temp, dims);
	
	checkCudaErrors(hipHostFree(temp));
}
